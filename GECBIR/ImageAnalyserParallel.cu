#include "hip/hip_runtime.h"
#include "ImageAnalyserParallel.h"


namespace GECBIR{



uchar3 * allocatecudaMemoryUchar3(uchar3 * hostdata, unsigned int dataSize, bool copyMemory)
{
	uchar3 *dev_a = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_a, dataSize * sizeof(uchar3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		return 0;
    }

	if(copyMemory)
	{
	cudaStatus = hipMemcpy(dev_a, hostdata, dataSize * sizeof(uchar3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 0;
    }
	}

    if(cudaStatus==hipSuccess)
		return dev_a;

	return 0;
}



int3 * allocatecudaMemoryInt3(int3 * hostdata, unsigned int dataSize, bool copyMemory)
{
	int3 *dev_a = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_a, dataSize * sizeof(int3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		return 0;
    }

	if(copyMemory)
	{
	cudaStatus = hipMemcpy(dev_a, hostdata, dataSize * sizeof(int3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 0;
    }
	}

    if(cudaStatus==hipSuccess)
		return dev_a;

	return 0;
}



hipError_t InitializeDevice()
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
    }
	return cudaStatus;
}


__global__ void histo_kernel(uchar3* d_Imgdata, int3* d_histo , unsigned int data_size)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if(idx < data_size)
	{
	atomicAdd(&(d_histo[d_Imgdata[idx].x].x),(int)1);
	atomicAdd(&(d_histo[d_Imgdata[idx].y].y),(int)1);
	atomicAdd(&(d_histo[d_Imgdata[idx].z].z),(int)1);
	}
	__syncthreads();

}

__global__ void histo_equal(uchar3 * d_thisImage, uchar3 * d_otherImage, bool * d_equalPixels, unsigned int data_size )
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if(idx<data_size)
	{
		if(d_thisImage[idx].x ==  d_otherImage[idx].x && d_thisImage[idx].y ==  d_otherImage[idx].y && d_thisImage[idx].z ==  d_otherImage[idx].z  )
			d_equalPixels[idx] = 1;
		else 
			d_equalPixels[idx] = 0;
	}
}

double3 findBhattacharyaDistance(int3 * hist1, int3 * hist2)
{
	int3 sum1 ={0,0,0};
	int3 sum2 = {0,0,0}; 

	for(int i =0; i<HISTOGRAM_BINS_SIZE; i++)
	{
		sum1.x += hist1[i].x ;
		sum1.y += hist1[i].y;
		sum1.z += hist1[i].z;

		sum2.x += hist2[i].x;
		sum2.y += hist2[i].y;
		sum2.z += hist2[i].z;
	}

	double3 mean1, mean2;

	mean1.x = sum1.x/(double)HISTOGRAM_BINS_SIZE ;
	mean1.y = sum1.y/(double)HISTOGRAM_BINS_SIZE ;
	mean1.z = sum1.z/(double)HISTOGRAM_BINS_SIZE ;

	mean2.x = sum2.x/(double)HISTOGRAM_BINS_SIZE ;
	mean2.y = sum2.y/(double)HISTOGRAM_BINS_SIZE ;
	mean2.z = sum2.z/(double)HISTOGRAM_BINS_SIZE ;

	double3 score = {0.0,0.0,0.0};

	for(int i =0; i<HISTOGRAM_BINS_SIZE; i++)
	{
		score.x += sqrt(hist1[i].x * hist2[i].x);
		score.y += sqrt(hist1[i].y * hist2[i].y);
		score.z += sqrt(hist1[i].z * hist2[i].z);
	}

	score.x = sqrt(1 - ( 1/ sqrt(mean1.x * mean2.x * HISTOGRAM_BINS_SIZE * HISTOGRAM_BINS_SIZE )) * score.x);
	score.y = sqrt(1 - ( 1/ sqrt(mean1.y * mean2.y * HISTOGRAM_BINS_SIZE * HISTOGRAM_BINS_SIZE )) * score.y);
	score.z = sqrt(1 - ( 1/ sqrt(mean1.z * mean2.z * HISTOGRAM_BINS_SIZE * HISTOGRAM_BINS_SIZE )) * score.z);

	return score;
}

int * ImageAnalyserParallel::ComputeHistogram()
{

	uchar3 * d_ImageData;
	int3 *d_histoData;


	int3 *histo_data;
	histo_data = new int3[HISTOGRAM_BINS_SIZE];
	for(int i =0; i<HISTOGRAM_BINS_SIZE ; i++)
	{
		histo_data[i].x = 0;
		histo_data[i].y = 0;
		histo_data[i].z = 0;
	}
	hipError_t cudaStatus;

	cudaStatus = InitializeDevice();
	if(cudaStatus != hipSuccess)
		return 0;

	d_ImageData = allocatecudaMemoryUchar3(this->PixelData,this->size_of_data,true);
	d_histoData = allocatecudaMemoryInt3(histo_data, HISTOGRAM_BINS_SIZE, true);  

	unsigned int data_size = this->size_of_data;

	int nThreads = HISTOGRAM_BINS_SIZE;
	int nBlocks = (data_size % nThreads == 0)?data_size/ nThreads: data_size/ nThreads+ 1;
	
	histo_kernel<<<nBlocks,nThreads>>>(d_ImageData, d_histoData, data_size ); 


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	
    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(histo_data, d_histoData, HISTOGRAM_BINS_SIZE * sizeof(int3), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	this->HistoData = histo_data;
	//double simi = findBhattacharyaDistance(histo_data, histo_data);

Error:
	hipFree(d_ImageData);
	hipFree(d_histoData);

	return 0;
}



ImageAnalyserParallel::ImageAnalyserParallel( uchar3 * data, int dataSize)
{
	PixelData = data;
	size_of_data = dataSize;
}



bool ImageAnalyserParallel::CompareImageSimilarity(ImageAnalyserParallel otherImage)
{
	otherImage.ComputeHistogram();
	double3 similarityScore =  findBhattacharyaDistance(this->HistoData, otherImage.HistoData);
	if(similarityScore.x < SIMILARITY_TOLERANCE_PARALLEL || similarityScore.x < SIMILARITY_TOLERANCE_PARALLEL || similarityScore.x < SIMILARITY_TOLERANCE_PARALLEL)
		return true;
	return false;
}








bool ImageAnalyserParallel::CompareImageEquality(ImageAnalyserParallel otherImage)
{
	uchar3 * d_thisImage;
	uchar3 * d_otherImage;
	bool * d_equalPixels;
	
	

	hipError_t cudaStatus;

	cudaStatus = InitializeDevice();
	if(cudaStatus != hipSuccess)
		return 0;

	unsigned int data_size = this->size_of_data;

	bool * equalPixels;
	equalPixels = new bool[data_size];

	d_thisImage = allocatecudaMemoryUchar3(this->PixelData,this->size_of_data,true);
	d_otherImage = allocatecudaMemoryUchar3(otherImage.PixelData,otherImage.size_of_data,true);

	cudaStatus = hipMalloc((void**)&d_equalPixels, data_size * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		return 0;
    }

	

	int nThreads = HISTOGRAM_BINS_SIZE;
	int nBlocks = (data_size % nThreads == 0)?data_size/ nThreads: data_size/ nThreads+ 1;
	
	histo_equal<<<nBlocks,nThreads>>>(d_thisImage, d_otherImage, d_equalPixels, data_size ); 


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	
    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(equalPixels, d_equalPixels, data_size * sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	long int sumEquals =0;
	for(int i =0 ;i<data_size ; i++)
	{
		sumEquals += equalPixels[i];
	}

	if(data_size - sumEquals <=20)
		return true;
	return false;

Error:
	hipFree(d_equalPixels);
	hipFree(d_thisImage);
	hipFree(d_otherImage);

	return 0;
}

}