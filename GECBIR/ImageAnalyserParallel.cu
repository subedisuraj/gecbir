#include "hip/hip_runtime.h"
#include "ImageAnalyserParallel.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>

namespace GECBIR{
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

int * allocatecudaMemory(const int * hostdata, unsigned int dataSize, bool copyMemory)
{
	int *dev_a = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_a, dataSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		return 0;
    }

	if(copyMemory)
	{
	cudaStatus = hipMemcpy(dev_a, hostdata, dataSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 0;
    }
	}

    if(cudaStatus==hipSuccess)
		return dev_a;

	return 0;
}


hipError_t InitializeDevice()
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
    }
	return cudaStatus;
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}



__global__ void histo_kernel(int *buffer, int size,  int *histo) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

// stride is total number of threads
    int stride = blockDim.x * gridDim.x;

 // All threads handle blockDim.x * gridDim.x
   // consecutive elements
   while (i < size) {
  /*    int alphabet_position = buffer[i] - 'a';
      if (alphabet_position >= 0 && alpha_position < 26) */		
	//	atomicAdd(&(histo[alphabet_position/4]), 1);
       i += stride;
   }
}



int addcuda()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


int * ImageAnalyserParallel::ComputeHistogram()
{
	int *d_ImageData;
	int *d_histoData;


	int *histo_data;
	hipError_t cudaStatus;

	cudaStatus = InitializeDevice();
	if(cudaStatus != hipSuccess)
		return 0;

	d_ImageData = allocatecudaMemory(this->PixelData,this->size_of_data,true);
	d_histoData = allocatecudaMemory(0,100, false);

	histo_kernel<<<1,this->size_of_data>>>(d_ImageData, this->size_of_data, d_histoData); 


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(histo_data, d_histoData, 100 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	hipFree(d_ImageData);
	hipFree(d_histoData);

	return 0;
}






// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = InitializeDevice();
	if(cudaStatus != hipSuccess)
		return cudaStatus;

	dev_c = allocatecudaMemory(c, size, false);
	dev_a = allocatecudaMemory(a, size, true);
	dev_b = allocatecudaMemory(b, size, true);


    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<< 1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}








ImageAnalyserParallel::ImageAnalyserParallel(int * data, int dataSize)
{
	PixelData = data;
	size_of_data = dataSize;
}



}