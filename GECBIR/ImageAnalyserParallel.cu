#include "hip/hip_runtime.h"
#include "ImageAnalyserParallel.h"

namespace GECBIR{

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

uchar3 * allocatecudaMemoryUchar3(uchar3 * hostdata, unsigned int dataSize, bool copyMemory)
{
	uchar3 *dev_a = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_a, dataSize * sizeof(uchar3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		return 0;
    }

	if(copyMemory)
	{
	cudaStatus = hipMemcpy(dev_a, hostdata, dataSize * sizeof(uchar3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 0;
    }
	}

    if(cudaStatus==hipSuccess)
		return dev_a;

	return 0;
}



int3 * allocatecudaMemoryInt3(int3 * hostdata, unsigned int dataSize, bool copyMemory)
{
	int3 *dev_a = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_a, dataSize * sizeof(int3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		return 0;
    }

	if(copyMemory)
	{
	cudaStatus = hipMemcpy(dev_a, hostdata, dataSize * sizeof(int3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 0;
    }
	}

    if(cudaStatus==hipSuccess)
		return dev_a;

	return 0;
}



hipError_t InitializeDevice()
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
    }
	return cudaStatus;
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}



__global__ void histo_kernel(uchar3* d_Imgdata, int3* d_histo , unsigned int data_size)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if(idx < data_size)
	{
	atomicAdd(&(d_histo[d_Imgdata[idx].x].x),(int)1);
	atomicAdd(&(d_histo[d_Imgdata[idx].y].y),(int)1);
	atomicAdd(&(d_histo[d_Imgdata[idx].z].z),(int)1);
	}
	__syncthreads();

}



double findBhattacharyaDistance(int3 * hist1, int3 * hist2)
{
	
//	int* data;
//int num;
//int3 x =  {0,0,0};
//thrust::device_vector< int3 > iVec(hist1,hist1+3);
//
//// transfer to device and compute sum
//int3 sum = thrust::reduce(iVec.begin(), iVec.end(), x , thrust::plus<int3>());
//int3 mean = sum/(double)num;
//
//   
//    float h2_R = mean(hist2);
//
//    
//    float score = 0;
//    for( int = 0; i< size(hist1); i++){
//        score += math.sqrt( hist1[i] * hist2[i] );
//	}
//    score = math.sqrt( 1 - ( 1 / math.sqrt(h1_*h2_*8*8) ) * score );
//    return score;



	int3 sum1 ={0,0,0}, sum2 = {0,0,0}; 
	for(int i =0; i<HISTOGRAM_BINS_SIZE; i++)
	{
		sum1.x += hist1[i].x ;
		sum1.y += hist1[i].y;
		sum1.z += hist1[i].z;

		sum2.x += hist2[i].x;
		sum2.y += hist2[i].y;
		sum2.z += hist2[i].z;
	}

	double3 mean1, mean2;

	mean1.x = sum1.x/(double)HISTOGRAM_BINS_SIZE ;
	mean1.y = sum1.y/(double)HISTOGRAM_BINS_SIZE ;
	mean1.z = sum1.z/(double)HISTOGRAM_BINS_SIZE ;

	mean2.x = sum1.x/(double)HISTOGRAM_BINS_SIZE ;
	mean2.y = sum1.y/(double)HISTOGRAM_BINS_SIZE ;
	mean2.z = sum1.z/(double)HISTOGRAM_BINS_SIZE ;

	double3 score = {0.0,0.0,0.0};

	for(int i =0; i<HISTOGRAM_BINS_SIZE; i++)
	{
		score.x += sqrt(hist1[i].x * hist2[i].x);
		score.y += sqrt(hist1[i].y * hist2[i].y);
		score.z += sqrt(hist1[i].z * hist2[i].z);

	}

	score.x = sqrt(1 - ( 1/ sqrt(mean1.x * mean2.x * HISTOGRAM_BINS_SIZE * HISTOGRAM_BINS_SIZE )) * score.x);
	score.y = sqrt(1 - ( 1/ sqrt(mean1.y * mean2.y * HISTOGRAM_BINS_SIZE * HISTOGRAM_BINS_SIZE )) * score.y);
	score.z = sqrt(1 - ( 1/ sqrt(mean1.z * mean2.z * HISTOGRAM_BINS_SIZE * HISTOGRAM_BINS_SIZE )) * score.z);


    return (score.x + score.y + score.z)/3;
}



int addcuda()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


int * ImageAnalyserParallel::ComputeHistogram()
{

	uchar3 * d_ImageData;
	int3 *d_histoData;


	int3 *histo_data;
	histo_data = new int3[HISTOGRAM_BINS_SIZE];
	hipError_t cudaStatus;

	cudaStatus = InitializeDevice();
	if(cudaStatus != hipSuccess)
		return 0;

	d_ImageData = allocatecudaMemoryUchar3(this->PixelData,this->size_of_data,true);
	d_histoData = allocatecudaMemoryInt3(histo_data, HISTOGRAM_BINS_SIZE * 3, false);  // 3 channels

	unsigned int data_size = this->size_of_data;

	int nThreads = 256;
	int nBlocks = data_size/ nThreads + 1;
	
	histo_kernel<<<nBlocks,nThreads>>>(d_ImageData, d_histoData, data_size ); 


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	
    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(histo_data, d_histoData, HISTOGRAM_BINS_SIZE * sizeof(int3), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	this->HistoData = histo_data;
	double simi = findBhattacharyaDistance(histo_data, histo_data);

Error:
	hipFree(d_ImageData);
	hipFree(d_histoData);

	return 0;
}






// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
return (hipError_t)0;
}
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//	cudaStatus = InitializeDevice();
//	if(cudaStatus != hipSuccess)
//		return cudaStatus;
//
//	dev_c = allocatecudaMemory(c, size, false);
//	dev_a = allocatecudaMemory(a, size, true);
//	dev_b = allocatecudaMemory(b, size, true);
//
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<< 1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}








ImageAnalyserParallel::ImageAnalyserParallel( uchar3 * data, int dataSize)
{
	PixelData = data;
	size_of_data = dataSize;
}



}