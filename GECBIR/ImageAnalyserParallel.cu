#include "hip/hip_runtime.h"
#include "ImageAnalyserParallel.h"

namespace GECBIR{

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

uchar3 * allocatecudaMemoryUchar3(uchar3 * hostdata, unsigned int dataSize, bool copyMemory)
{
	uchar3 *dev_a = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_a, dataSize * sizeof(uchar3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		return 0;
    }

	if(copyMemory)
	{
	cudaStatus = hipMemcpy(dev_a, hostdata, dataSize * sizeof(uchar3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 0;
    }
	}

    if(cudaStatus==hipSuccess)
		return dev_a;

	return 0;
}



int3 * allocatecudaMemoryInt3(int3 * hostdata, unsigned int dataSize, bool copyMemory)
{
	int3 *dev_a = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_a, dataSize * sizeof(int3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		return 0;
    }

	if(copyMemory)
	{
	cudaStatus = hipMemcpy(dev_a, hostdata, dataSize * sizeof(int3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 0;
    }
	}

    if(cudaStatus==hipSuccess)
		return dev_a;

	return 0;
}



hipError_t InitializeDevice()
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
    }
	return cudaStatus;
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}



__global__ void histo_kernel(uchar3* d_Imgdata, int3* d_histo , unsigned int data_size)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if(idx < data_size)
	{
	atomicAdd(&(d_histo[d_Imgdata[idx].x].x),(int)1);
	atomicAdd(&(d_histo[d_Imgdata[idx].y].y),(int)1);
	atomicAdd(&(d_histo[d_Imgdata[idx].z].z),(int)1);
	}
	__syncthreads();

}



int addcuda()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


int * ImageAnalyserParallel::ComputeHistogram()
{

	uchar3 * d_ImageData;
	int3 *d_histoData;


	int3 *histo_data;
	histo_data = new int3[HISTOGRAM_BINS_SIZE];
	hipError_t cudaStatus;

	cudaStatus = InitializeDevice();
	if(cudaStatus != hipSuccess)
		return 0;

	d_ImageData = allocatecudaMemoryUchar3(this->PixelData,this->size_of_data,true);
	d_histoData = allocatecudaMemoryInt3(histo_data, HISTOGRAM_BINS_SIZE * 3, false);  // 3 channels

	unsigned int data_size = this->size_of_data;

	int nThreads = 256;
	int nBlocks = data_size/ nThreads + 1;
	
	histo_kernel<<<nBlocks,nThreads>>>(d_ImageData, d_histoData, data_size ); 


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	
    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(histo_data, d_histoData, HISTOGRAM_BINS_SIZE * sizeof(int3), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	this->HistoData = histo_data;

Error:
	hipFree(d_ImageData);
	hipFree(d_histoData);

	return 0;
}






// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
return (hipError_t)0;
}
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//	cudaStatus = InitializeDevice();
//	if(cudaStatus != hipSuccess)
//		return cudaStatus;
//
//	dev_c = allocatecudaMemory(c, size, false);
//	dev_a = allocatecudaMemory(a, size, true);
//	dev_b = allocatecudaMemory(b, size, true);
//
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<< 1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}








ImageAnalyserParallel::ImageAnalyserParallel( uchar3 * data, int dataSize)
{
	PixelData = data;
	size_of_data = dataSize;
}



}